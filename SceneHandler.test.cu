#include "hip/hip_runtime.h"
#include "SceneHandler.test.cuh"
#include "SceneHandler.cuh"
#include "MeshReader.test.h"
#include "SimpleDirectionalLight.cuh"
#include "DefaultPerspectiveLense.cuh"
#include "Tests.h"
#include <iostream>
#include <thread>
#include <mutex>
#include "Semaphore.h"



namespace SceneHandlerTest {
	namespace Private{
		__global__ void kernel(const Scene<BakedTriFace> *scene) {
			Vector2 screenPoint = ((Vector2(threadIdx.x, blockIdx.x) / Vector2(blockDim.x, gridDim.x)) - Vector2(0.5f, 0.0f));
			scene->geometry.cast(scene->cameras[0].lense.getScreenPhoton(screenPoint).ray);
		}

		static void runKernels(const volatile bool *quit, const SceneHandler<BakedTriFace> *scene, int index, std::mutex *ioLock, Semaphore *initSem, std::mutex *exitLock) {
			if (!scene->selectGPU(index)) {
				ioLock->lock();
				std::cout << "Thread " << index << " could not select the GPU." << std::endl;
				ioLock->unlock();
				initSem->post();
				return;
			}
			if (scene->getHandleGPU(index) == NULL) {
				ioLock->lock();
				std::cout << "GPU handle missing for thread " << index << "." << std::endl;
				ioLock->unlock();
				initSem->post();
				return;
			}
			ioLock->lock();
			std::cout << "Thread " << index << " active." << std::endl;
			ioLock->unlock();
			initSem->post();
			while (true) {
				if (*quit) break;
				kernel<<<256,256>>>(scene->getHandleGPU(index));
				if (hipDeviceSynchronize() != hipSuccess) {
					exitLock->lock();
					ioLock->lock();
					std::cout << "Thread " << index << " failed." << std::endl;
					ioLock->unlock();
					exitLock->unlock();
					return;
				}
			}
			ioLock->lock();
			std::cout << "Thread " << index << " quit." << std::endl;
			ioLock->unlock();
		}
		static void makeScene(Scene<BakedTriFace> &scene) {
			Stacktor<PolyMesh> meshes; 
			MeshReaderTest::readMeshes(meshes);
			for (int i = 0; i < meshes.size(); i++) {
				scene.geometry.push(meshes[i].bake());
				std::cout << "\rPUSHED " << i;
			}
			std::cout << std::endl << "BUILDING. PLEASE WAIT..." << std::endl;
			scene.geometry.build();
			scene.lights.flush(1);
			Vector3 direction = Vector3(0.2f, -0.4f, 0.7f).normalized();
			scene.lights[0].use<SimpleDirectionalLight>(Photon(Ray(-direction * 10000.0f, direction), Color(1.0f, 1.0f, 1.0f)));
			scene.cameras.flush(1);
			scene.cameras[0].transform.setPosition(Vector3(0, 0, -128));
			scene.cameras[0].lense.use<DefaultPerspectiveLense>(60.0f);
		}
		static void runTest() {
			Scene<BakedTriFace> scene;
			makeScene(scene);
			SceneHandler<BakedTriFace> handler(scene);
			std::cout << "Uploading..." << std::endl;
			handler.uploadToEveryGPU();
			volatile bool quit = false;
			std::mutex ioLock;
			Semaphore initSem;
			std::mutex exitLock;
			exitLock.lock();
			std::thread *threads = new std::thread[handler.gpuCount()];
			for (int i = 0; i < handler.gpuCount(); i++)
				threads[i] = std::thread(runKernels, &quit, &handler, i, &ioLock, &initSem, &exitLock);
			for (int i = 0; i < handler.gpuCount(); i++) initSem.wait();
			ioLock.lock();
			std::cout << "Should likely be running some kernels on GPU-s. Enter anthing to quit... ";
			ioLock.unlock();
			std::string s;
			std::getline(std::cin, s);
			quit = true;
			exitLock.unlock();
			for (int i = 0; i < handler.gpuCount(); i++) threads[i].join();
			delete[] threads;
		}
		static void test() {
			while (true) {
				std::cout << "Enter anthing to run SceneHandler test: ";
				std::string s;
				std::getline(std::cin, s);
				if (s.length() <= 0) break;
				runTest();
			}
		}
	}

	void test() {
		Tests::runTest(Private::test, "Testing SceneHandler");
		hipSetDevice(0);
	}
}



