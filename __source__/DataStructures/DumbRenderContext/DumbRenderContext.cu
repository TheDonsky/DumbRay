#include "hip/hip_runtime.h"
#include "DumbRenderContext.cuh"
#include "../../Namespaces/MeshReader/MeshReader.h"
#include "../Screen/FrameBuffer/BlockBasedFrameBuffer/BlockBasedFrameBuffer.cuh"
#include "../Objects/Components/Lenses/SimpleStochasticLense/SimpleStochasticLense.cuh"
#include "../../Namespaces/Images/Images.cuh"
#include "DumbRenderContextRegistry.cuh"
#include <fstream>
#include <sstream>
#include <iomanip>
#include <vector>
#include <mutex>
#include <set>

namespace {
	static std::mutex registryLock;
	static DumbRenderContextRegistry *registry = NULL;
	
	typedef std::unordered_map<std::string, DumbRenderContextRegistry::MaterialFromDsonFunction> MaterialParserMap;
	typedef std::unordered_map<std::string, DumbRenderContextRegistry::LightFromDsonFunction> LightParserMap;
	typedef std::unordered_map<std::string, DumbRenderContextRegistry::LenseFromDsonFunction> LenseParserMap;

	MaterialParserMap materialParsers;
	LightParserMap lightParsers;
	LenseParserMap lenseParsers;

	struct DumbRenderContextData;

	typedef std::unordered_map<std::string, PolyMesh> MeshDict;
	typedef std::unordered_map<std::string, MeshDict> ObjDict;

	struct DumbRenderContextData {
		std::string sourcePath;

		std::unordered_map<std::string, int> materials;
		std::unordered_map<std::string, int> textures;

		ObjDict objectFiles;

		DumbRenderer::SceneType scene;
		ReferenceManager<Camera> camera;

		Renderer::ThreadConfiguration threadConfiguration;
		BlockRenderer::BlockConfiguration blockConfiguration;
		struct RendererSettings {
			DumbRenderer::BoxingMode boxingMode;
			int maxBounces;
			int samplesPerPixelX, samplesPerPixelY;
			int pixelsPerGPUThread;
			bool ignoreBackfaces;
		};
		RendererSettings rendererSettings;

		struct Group {
			struct Item {
				std::string fileName;
				std::string objectName;
				Transform transform;
				int materialId;

				inline Item() { materialId = -1; }
				inline Stacktor<DumbRenderer::SceneType::GeometryUnit> resolve(const DumbRenderContextData *data)const;
				inline bool fromDson(const Dson::Object *object, std::ostream *errorStream, DumbRenderContextData *data);
			};
			struct SubGroup {
				std::string group;
				Transform transform;
				int fallbackMaterial;
				int overrideMaterial;

				inline SubGroup() { fallbackMaterial = overrideMaterial = -1; }
				inline Stacktor<DumbRenderer::SceneType::GeometryUnit> resolve(const DumbRenderContextData *data)const;
				inline bool fromDson(const Dson::Object *object, std::ostream *errorStream, DumbRenderContextData *data);
			};
			std::string name;
			std::vector<Item> groupItems;
			std::vector<SubGroup> subGroups;
			Transform transform;
			int fallbackMaterial;
			int overrideMaterial;

			inline Group() { fallbackMaterial = overrideMaterial = -1; }
			inline Stacktor<DumbRenderer::SceneType::GeometryUnit> resolve(const DumbRenderContextData *data)const;
			inline bool fromDson(const Dson::Object *object, std::ostream *errorStream, DumbRenderContextData *data);
		};

		std::map<std::string, Group> groups;

		bool parseMaterials(const Dson::Object &object, std::ostream *errorStream);
		bool parseLights(const Dson::Object &object, std::ostream *errorStream);
		bool parseGroups(const Dson::Object &object, std::ostream *errorStream);
		bool parseObjects(const Dson::Object &object, std::ostream *errorStream);
		bool parseCamera(const Dson::Object &object, std::ostream *errorStream);
		bool parseRenderer(const Dson::Object &object, std::ostream *errorStream);
		bool includeFiles(const Dson::Object &object, std::ostream *errorStream);

		bool parseMaterial(const Dson::Object &object, std::ostream *errorStream, int *materialId = NULL);
		bool parseLight(const Dson::Object &object, std::ostream *errorStream);
		bool parseObject(const Dson::Object &object, std::ostream *errorStream);

		bool loadObjFile(const std::string &filename, std::ostream *errorStream);
		bool getMaterialId(const Dson::Object &object, std::ostream *errorStream, int &materialId);

		void *data;
		DumbRenderContext *owner;
		inline DumbRenderContextData(DumbRenderContext *o) { data = this; owner = o; }
	};

	inline Stacktor<DumbRenderer::SceneType::GeometryUnit> DumbRenderContextData::Group::Item::resolve(const DumbRenderContextData *data)const {
		Stacktor<DumbRenderer::SceneType::GeometryUnit> resolved;
		if (objectName != "") {
			const BakedTriMesh mesh = data->objectFiles.find(fileName)->second.find(objectName)->second.bake();
			for (int i = 0; i < mesh.size(); i++)
				resolved.push(DumbRenderer::SceneType::GeometryUnit(mesh[i] >> transform, materialId));
		}
		else {
			const MeshDict &meshes = data->objectFiles.find(fileName)->second;
			for (MeshDict::const_iterator it = meshes.begin(); it != meshes.end(); it++) {
				const BakedTriMesh mesh = it->second.bake();
				for (int i = 0; i < mesh.size(); i++)
					resolved.push(DumbRenderer::SceneType::GeometryUnit(mesh[i] >> transform, materialId));
			}
		}
		return resolved;
	}

	inline Stacktor<DumbRenderer::SceneType::GeometryUnit> DumbRenderContextData::Group::SubGroup::resolve(const DumbRenderContextData *data)const {
		Stacktor<DumbRenderer::SceneType::GeometryUnit> resolved = data->groups.find(group)->second.resolve(data);
		for (int i = 0; i < resolved.size(); i++) {
			if (overrideMaterial >= 0) resolved[i].materialId = overrideMaterial;
			else if (resolved[i].materialId < 0) resolved[i].materialId = fallbackMaterial;
			resolved[i].object >>= transform;
		}
		return resolved;
	}

	inline Stacktor<DumbRenderer::SceneType::GeometryUnit> DumbRenderContextData::Group::resolve(const DumbRenderContextData *data)const {
		Stacktor<DumbRenderer::SceneType::GeometryUnit> resolved;
		for (size_t i = 0; i < groupItems.size(); i++) {
			Stacktor<DumbRenderer::SceneType::GeometryUnit> addition = groupItems[i].resolve(data);
			for (int j = 0; j < addition.size(); j++) resolved.push(addition[j]);
		}
		for (size_t i = 0; i < subGroups.size(); i++) {
			Stacktor<DumbRenderer::SceneType::GeometryUnit> addition = subGroups[i].resolve(data);
			for (int j = 0; j < addition.size(); j++) resolved.push(addition[j]);
		}
		for (int i = 0; i < resolved.size(); i++) {
			if (overrideMaterial >= 0) resolved[i].materialId = overrideMaterial;
			else if (resolved[i].materialId < 0) resolved[i].materialId = fallbackMaterial;
			resolved[i].object >>= transform;
		}
		return resolved;
	}

	inline bool DumbRenderContextData::Group::Item::fromDson(const Dson::Object *object, std::ostream *errorStream, DumbRenderContextData *data) {
		const Dson::Dict *dict = object->safeConvert<Dson::Dict>(errorStream, "Error: Group item can only be made from a dictionary"); if (dict == NULL) return false;
		if (dict->contains("mesh")) {
			const Dson::Dict *meshDict = dict->get("mesh").safeConvert<Dson::Dict>(errorStream, "Error: Item 'mesh' entry has to be a dict"); if (meshDict == NULL) return false;
			if (meshDict->contains("obj")) {
				const Dson::String *objFileName = meshDict->get("obj").safeConvert<Dson::String>(errorStream, "Error: .obj file name must be a string"); if (objFileName == NULL) return false;
				if (!data->loadObjFile(objFileName->value(), errorStream)) return false;
				fileName = ("obj::" + objFileName->value());
			}
			else if (meshDict->contains("primitive")) {
				// __TODO__: create a primitive...
			}
			else {
				if (errorStream != NULL) (*errorStream) << "Error: Object mesh could not be parsed" << std::endl;
				return false;
			}

			if (meshDict->contains("object")) {
				const Dson::String *meshName = meshDict->get("object").safeConvert<Dson::String>(errorStream, "Error: .obj mesh name must be a string"); if (meshName == NULL) return false;
				const MeshDict &fileMeshes = data->objectFiles[fileName];
				if (fileMeshes.find(meshName->value()) == fileMeshes.end()) {
					if (errorStream != NULL) (*errorStream) << "Error: Object [" + meshName->value() + "] not found in [" + fileName + "]\n";
					return false;
				}
				objectName = meshName->value();
			}
			else objectName = "";
		}
		else {
			if (errorStream != NULL) (*errorStream) << "Error: Item has to have a 'mesh' entry" << std::endl;
			return false;
		}

		if (dict->contains("transform")) if (!transform.fromDson(dict->get("transform"), errorStream)) return false;

		if (dict->contains("material")) if (!data->getMaterialId(dict->get("material"), errorStream, materialId)) return false;

		return true;
	}
	inline bool DumbRenderContextData::Group::SubGroup::fromDson(const Dson::Object *object, std::ostream *errorStream, DumbRenderContextData *data) {
		if (object->type() == Dson::Object::DSON_STRING) {
			group = ("named::" + ((const Dson::String*)object)->value());
			if (data->groups.find(group) == data->groups.end()) {
				if (errorStream != NULL) (*errorStream) << ("Error: Group [" + ((Dson::String*)object)->value() + "] not found\n");
				return false;
			}
		}
		else {
			const Dson::Dict *groupDict = object->safeConvert<Dson::Dict>(errorStream, "Error: Subgroup should be expressed either as a string, or a dict"); if (groupDict == NULL) return false;
			if (groupDict->contains("group")) {
				const Dson::String *groupName = groupDict->get("group").safeConvert<Dson::String>(errorStream, "Error: Subgroup with pre-defined group reference must have 'group' key, paired with string value"); if (groupName == NULL) return false;
				group = ("named::" + groupName->value());
				if (data->groups.find(group) == data->groups.end()) {
					if (errorStream != NULL) (*errorStream) << ("Error: Group [" + group + "] not found\n");
					return false;
				}
				if (groupDict->contains("transform")) if (!transform.fromDson(groupDict->get("transform"), errorStream)) return false;
				if (groupDict->contains("fallback_material")) if (!data->getMaterialId(groupDict->get("fallback_material"), errorStream, fallbackMaterial)) return false;
				if (groupDict->contains("override_material")) if (!data->getMaterialId(groupDict->get("override_material"), errorStream, overrideMaterial)) return false;
			}
			else {
				Group subGroup; if (!subGroup.fromDson(groupDict, errorStream, data)) return false;
				group = subGroup.name;
			}
		}
		return true;
	}
	inline bool DumbRenderContextData::Group::fromDson(const Dson::Object *object, std::ostream *errorStream, DumbRenderContextData *data) {
		const Dson::Dict *dict = object->safeConvert<Dson::Dict>(errorStream, "Error: Group can only be made from a dictionary"); if (dict == NULL) return false;
		if (dict->contains("elements")) {
			const Dson::Array *elements = dict->get("elements").safeConvert<Dson::Array>(errorStream, "Error: Group elements should be contained in an array"); if (elements == NULL) return false;
			for (int i = 0; i < elements->size(); i++) {
				const Dson::Object &elementObject = elements->get(i);
				const Dson::Dict *elementDict = elementObject.safeConvert<Dson::Dict>();
				if (elementDict != NULL && elementDict->contains("mesh")) {
					Item item; if (!item.fromDson(&elementObject, errorStream, data)) return false;
					groupItems.push_back(item);
				}
				else {
					SubGroup group; if (!group.fromDson(&elementObject, errorStream, data)) return false;
					subGroups.push_back(group);
				}
			}
		}
		
		if (dict->contains("transform")) if (!transform.fromDson(dict->get("transform"), errorStream)) return false;
		
		if (dict->contains("fallback_material")) if (!data->getMaterialId(dict->get("fallback_material"), errorStream, fallbackMaterial)) return false;
		if (dict->contains("override_material")) if (!data->getMaterialId(dict->get("override_material"), errorStream, overrideMaterial)) return false;
		
		if (dict->contains("name")) {
			const Dson::String *nameObject = dict->get("name").safeConvert<Dson::String>(errorStream, "Error: Group name must be a string"); if (nameObject == NULL) return false;
			name = ("named::" + nameObject->value());
		}
		else {
			std::stringstream stream;
			name = ("unnamed::" + ((std::stringstream*)(&(stream << (data->groups.size()))))->str());
		}
		data->groups[name] = (*this);
		return true;
	}
}


void DumbRenderContextRegistry::registerMaterialType(
	const std::string &typeName, MaterialFromDsonFunction fromDsonFunction) {
	materialParsers[typeName] = fromDsonFunction;
}
void DumbRenderContextRegistry::registerLightType(
	const std::string &typeName, LightFromDsonFunction fromDsonFunction) {
	lightParsers[typeName] = fromDsonFunction;
}
void DumbRenderContextRegistry::registerLenseType(
	const std::string &typeName, LenseFromDsonFunction fromDsonFunction) {
	lenseParsers[typeName] = fromDsonFunction;
}


#define CONTEXT ((DumbRenderContextData*)data)->

DumbRenderContext::DumbRenderContext() {
	{
		std::lock_guard<std::mutex> guard(registryLock);
		if (registry == NULL) registry = new DumbRenderContextRegistry();
	}
	DumbRenderContextData *dataObject = new DumbRenderContextData(this);
	data = ((void*)dataObject);
	if (data == NULL) return;
	dataObject->threadConfiguration = Renderer::ThreadConfiguration(Renderer::ThreadConfiguration::ALL_BUT_GPU_THREADS, 2);
	dataObject->rendererSettings.boxingMode = DumbRenderer::BOXING_MODE_HEIGHT_BASED;
	dataObject->rendererSettings.maxBounces = 2;
	dataObject->rendererSettings.samplesPerPixelX = 1;
	dataObject->rendererSettings.samplesPerPixelY = 1;
	dataObject->rendererSettings.pixelsPerGPUThread = 1;
	dataObject->rendererSettings.ignoreBackfaces = true;
}
DumbRenderContext::~DumbRenderContext() {
	DumbRenderContextData *dataObject = ((DumbRenderContextData*)data);
	if (dataObject != NULL) { delete dataObject; data = NULL; }
}

bool DumbRenderContext::buildFromFile(const std::string &filename, std::ostream *errorStream) {
	this->~DumbRenderContext();
	new (this) DumbRenderContext();
	if (!fromFile(filename, errorStream)) return false;
	CONTEXT scene.geometry.cpuHandle()->build();
	return true;
}
bool DumbRenderContext::buildFromDson(const Dson::Object *object, std::ostream *errorStream) {
	this->~DumbRenderContext();
	new (this) DumbRenderContext();
	if (!fromDson(object, errorStream)) return false;
	CONTEXT scene.geometry.cpuHandle()->build();
	return true;
}

bool DumbRenderContext::fromFile(const std::string &filename, std::ostream *errorStream) {
	std::ifstream stream;
	stream.open(filename.c_str());
	std::string string((std::istreambuf_iterator<char>(stream)), std::istreambuf_iterator<char>());
	if (stream.fail()) {
		if (errorStream != NULL) (*errorStream) << ("Error: Could not open file: \"" + filename + "\"..") << std::endl;
		return false;
	}
	Dson::Object *object = Dson::parse(string, errorStream);
	if (object != NULL) {
		const std::string src = CONTEXT sourcePath;
		{
			size_t len = (filename.length());
			while ((len <= filename.length()) && (len > 0) && (filename[len - 1] != '/') && (filename[len - 1] != '\\')) len--;
			CONTEXT sourcePath = filename.substr(0, len);
		}
		bool rv = fromDson(object, errorStream);
		CONTEXT sourcePath = src;
		delete object;
		return rv;
	}
	else {
		if (errorStream != NULL) (*errorStream) << "Error: Could not parse file: \"" << filename << "\"" << std::endl;
		return false;
	}
}

bool DumbRenderContext::fromDson(const Dson::Object *object, std::ostream *errorStream) {
	if (object == NULL) {
		if (errorStream != NULL) (*errorStream) << "Error: Render context can not be constructed from a NULL Dson::Object" << std::endl;
		return false;
	}
	if (object->type() != Dson::Object::DSON_DICT) {
		if (errorStream != NULL) (*errorStream) << "Error: Render context can be constructed only from a Dson::Dict" << std::endl;
		return false;
	}
	const Dson::Dict &dict = (*((const Dson::Dict*)object));
	if (dict.contains("include")) {
		if (!CONTEXT includeFiles(dict.get("include"), errorStream)) return false;
	}
	if (dict.contains("materials")) {
		if (!CONTEXT parseMaterials(dict.get("materials"), errorStream)) return false;
	}
	if (dict.contains("lights")) {
		if (!CONTEXT parseLights(dict.get("lights"), errorStream)) return false;
	}
	if (dict.contains("groups")) {
		if (!CONTEXT parseGroups(dict.get("groups"), errorStream)) return false;
	}
	if (dict.contains("objects")) {
		if (!CONTEXT parseObjects(dict.get("objects"), errorStream)) return false;
	}
	if (dict.contains("camera")) {
		if (!CONTEXT parseCamera(dict.get("camera"), errorStream)) return false;
	}
	if (CONTEXT camera.cpuHandle()->lense.object() == NULL)
		CONTEXT camera.cpuHandle()->lense.use<SimpleStochasticLense>(64.0f);
	if (dict.contains("renderer")) {
		if (!CONTEXT parseRenderer(dict.get("renderer"), errorStream)) return false;
	}
	return true;
}


bool DumbRenderContext::getImageId(const Dson::Object &object, int *imageId, std::ostream *errorStream) {
	if ((object.type() == Dson::Object::DSON_BOOL)
		|| (object.type() == Dson::Object::DSON_NULL)
		|| (object.type() == Dson::Object::DSON_NUMBER)) {
		(*imageId) = (-1);
	}
	else if (object.type() == Dson::Object::DSON_STRING) {
		const std::string &name = ((const Dson::String*)(&object))->value();
		std::unordered_map<std::string, int>::const_iterator it = CONTEXT textures.find("name::" + name);
		if (it != CONTEXT textures.end()) { (*imageId) = it->second; return true; }
		it = CONTEXT textures.find("png::" + name);
		if (it != CONTEXT textures.end()) { (*imageId) = it->second; return true; }

		if (errorStream != NULL) (*errorStream) << ("Error: Texture \"" + name + "\" not found") << std::endl;
		return false;
	}
	else if (object.type() == Dson::Object::DSON_DICT) {
		const Dson::Dict &dict = (*((const Dson::Dict*)(&object)));
		if (dict.contains("png")) {
			const Dson::String *fileNameObject = dict.get("png").safeConvert<Dson::String>(errorStream, "Error: Image 'png' entry MUST BE a string");
			if (fileNameObject == NULL) return false;
			const std::string &fileName = fileNameObject->value();
			std::string filePath = (CONTEXT sourcePath + fileName);
			{
				std::ifstream stream;
				stream.open(filePath);
				if (stream.fail()) filePath = fileName;
			}
			Texture texture;
			if (Images::getTexturePNG(texture, filePath) == Images::IMAGES_NO_ERROR) {
				(*imageId) = CONTEXT scene.textures.cpuHandle()->size();
				CONTEXT scene.textures.cpuHandle()->flush(1);
				CONTEXT scene.textures.cpuHandle()->operator[](*imageId).stealFrom(texture);
				CONTEXT textures["png::" + filePath] = (*imageId);
				CONTEXT textures["png::" + fileName] = (*imageId);
			}
			else {
				if (errorStream != NULL) (*errorStream) << ("Error: Could not read file: \"" + filePath + "\"") << std::endl;
				return false;
			}
		}
		// MAYBE... ADD OPTIONS TO ADD SOME OTHER WAYS TO GENERATE IMAGES....
		else {
			if (errorStream != NULL) (*errorStream) << "Error: Image dict incomplete" << std::endl;
			return false;
		}

		if (dict.contains("filtering")) {
			const Dson::String *filterObject = dict.get("filtering").safeConvert<Dson::String>(errorStream, "Error: Image 'filtering' entry MUST BE a string");
			if (filterObject == NULL) return false;
			const std::string &filter = filterObject->value();
			if (filter == "none") CONTEXT scene.textures.cpuHandle()->operator[](*imageId).setFiltering(Texture::FILTER_NONE);
			else if (filter == "bilinear") CONTEXT scene.textures.cpuHandle()->operator[](*imageId).setFiltering(Texture::FILTER_BILINEAR);
			else {
				if (errorStream != NULL) (*errorStream) << ("Error: Image filter can be only \"none\"/[\"bilinear\"] (got: \"" + filter + "\")") << std::endl;
				return false;
			}
		}

		if (dict.contains("name")) {
			const Dson::String *nameObject = dict.get("name").safeConvert<Dson::String>(errorStream, "Error: Image name entry MUST BE a string");
			if (nameObject == NULL) return false;
			CONTEXT textures["name::" + nameObject->value()] = (*imageId);
		}
	}
	else {
		if (errorStream != NULL) (*errorStream) << "Error: Unsupported dson type for texture" << std::endl;
		return false;
	}
	return true;
}





bool DumbRenderContextData::parseMaterials(const Dson::Object &object, std::ostream *errorStream) {
	if (object.type() != Dson::Object::DSON_ARRAY) {
		if (errorStream != NULL) (*errorStream) << "Error: Materials should be contained in Dson::Array" << std::endl;
		return false;
	}
	const Dson::Array &arr = (*((Dson::Array*)(&object)));
	for (size_t i = 0; i < arr.size(); i++)
		if (!parseMaterial(arr[i], errorStream)) return false;
	return true;
}
bool DumbRenderContextData::parseLights(const Dson::Object &object, std::ostream *errorStream) {
	if (object.type() != Dson::Object::DSON_ARRAY) {
		if (errorStream != NULL) (*errorStream) << "Error: Lights should be contained in Dson::Array" << std::endl;
		return false;
	}
	const Dson::Array &arr = (*((Dson::Array*)(&object)));
	for (size_t i = 0; i < arr.size(); i++)
		if (!parseLight(arr[i], errorStream)) return false;
	return true;
}
bool DumbRenderContextData::parseGroups(const Dson::Object &object, std::ostream *errorStream) {
	if (object.type() != Dson::Object::DSON_ARRAY) {
		if (errorStream != NULL) (*errorStream) << "Error: Groups should be contained in Dson::Array" << std::endl;
		return false;
	}
	const Dson::Array &arr = (*((Dson::Array*)(&object)));
	for (size_t i = 0; i < arr.size(); i++)
		if (!Group().fromDson(&arr[i], errorStream, this)) return false;
	return true;
}
bool DumbRenderContextData::parseObjects(const Dson::Object &object, std::ostream *errorStream) {
	if (object.type() != Dson::Object::DSON_ARRAY) {
		if (errorStream != NULL) (*errorStream) << "Error: Objects should be contained in Dson::Array" << std::endl;
		return false;
	}
	const Dson::Array &arr = (*((Dson::Array*)(&object)));
	for (size_t i = 0; i < arr.size(); i++)
		if (!parseObject(arr[i], errorStream)) return false;
	return true;
}
bool DumbRenderContextData::parseCamera(const Dson::Object &object, std::ostream *errorStream) {
	if (object.type() != Dson::Object::DSON_DICT) {
		if (errorStream != NULL) (*errorStream) << "Error: Camera should be contained in Dson::Dict" << std::endl;
		return false;
	}
	const Dson::Dict &dict = (*((Dson::Dict*)(&object)));
	if (dict.contains("lense")) {
		const Dson::Object &lenseObject = dict.get("lense");
		if (lenseObject.type() != Dson::Object::DSON_DICT) {
			if (errorStream != NULL) (*errorStream) << "Error: Camera lense has to be a Dson::Dict" << std::endl;
			return false;
		}
		const Dson::Dict &lense = (*((Dson::Dict*)(&lenseObject)));
		if (!lense.contains("type")) {
			if (errorStream != NULL) (*errorStream) << "Error: Lense has to have a type" << std::endl;
			return false;
		}
		else {
			const Dson::Object &typeObject = lense.get("type");
			if (typeObject.type() != Dson::Object::DSON_STRING) {
				if (errorStream != NULL) (*errorStream) << "Error: Lense type has to be a string" << std::endl;
				return false;
			}
			const std::string &type = ((Dson::String*)(&typeObject))->value();
			LenseParserMap::const_iterator it = lenseParsers.find(type);
			if (it == lenseParsers.end()) {
				if (errorStream != NULL) (*errorStream) << "Error: Unknown lense type: \"" << type << "\"" << std::endl;
				return false;
			}
			Lense tmpLense;
			if (!it->second(tmpLense, lense, errorStream, this->owner)) return false;
			CONTEXT camera.cpuHandle()->lense = tmpLense;
		}
	}
	if (dict.contains("transform")) {
		Transform transform;
		if (!transform.fromDson(dict["transform"], errorStream)) return false;
		CONTEXT camera.cpuHandle()->transform = transform;
	}
	return true;
}
bool DumbRenderContextData::parseRenderer(const Dson::Object &object, std::ostream *errorStream) {
	const Dson::Dict *renderer = object.safeConvert<Dson::Dict>(errorStream, "Error: Renderer should be contained in Dson::Dict");
	if (renderer == NULL) return false;

	if (renderer->contains("resources")) {
		const Dson::Dict *resources = renderer->get("resources").safeConvert<Dson::Dict>(errorStream, "Error: Renderer Resources has to be a Dson::Dict type");
		if (resources == NULL) return false;
		if (resources->contains("cpu")) {
			const Dson::Object &cpuObject = resources->get("cpu");
			if (cpuObject.type() == Dson::Object::DSON_STRING) {
				const std::string &cpuText = ((const Dson::String*)(&cpuObject))->value();
				if (cpuText == "all") CONTEXT threadConfiguration.configureCPU(Renderer::ThreadConfiguration::ALL);
				else if (cpuText == "all_but_gpu") CONTEXT threadConfiguration.configureCPU(Renderer::ThreadConfiguration::ALL_BUT_GPU_THREADS);
				else if (cpuText == "all_but_one_per_gpu") CONTEXT threadConfiguration.configureCPU(Renderer::ThreadConfiguration::ALL_BUT_THREAD_PER_GPU);
				else if (cpuText == "none") CONTEXT threadConfiguration.configureCPU(Renderer::ThreadConfiguration::NONE);
				else if (cpuText == "one") CONTEXT threadConfiguration.configureCPU(Renderer::ThreadConfiguration::ONE);
				else {
					if (errorStream != NULL) (*errorStream) << "Error: Renderer Resources CPU can only be one of 'all'/['all_but_gpu']/'all_but_one_per_gpu'/'none'/'one'/any_number" << std::endl;
					return false;
				}
			}
			else if (cpuObject.type() == Dson::Object::DSON_NUMBER)
				CONTEXT threadConfiguration.configureCPU(((const Dson::Number*)(&cpuObject))->intValue());
			else {
				if (errorStream != NULL) (*errorStream) << "Error: Renderer Resources CPU can only be one of 'all'/['all_but_gpu']/'all_but_one_per_gpu'/'none'/'one'/any_number" << std::endl;
				return false;
			}
		}
		if (resources->contains("gpu")) {
			const Dson::Number *gpu = resources->get("gpu").safeConvert<Dson::Number>(errorStream, "Error: Renderer Resources GPU can only be a number");
			if (gpu == NULL) return false;
			CONTEXT threadConfiguration.configureEveryGPU(gpu->intValue());
		}
		for (int i = 0; i < CONTEXT threadConfiguration.numDevices(); i++) {
			std::stringstream stream;
			stream << "gpu_" << i;
			const std::string &gpuKey = stream.str();
			if (resources->contains(gpuKey)) {
				const Dson::Number *gpu = resources->get(gpuKey).safeConvert<Dson::Number>(errorStream, "Error: Renderer Resources " + gpuKey + " can only be a number");
				if (gpu == NULL) return false;
				CONTEXT threadConfiguration.configureGPU(i, gpu->intValue());
			}
		}
	}
	
	if (renderer->contains("blocks")) {
		const Dson::Dict *blocks = renderer->get("blocks").safeConvert<Dson::Dict>(errorStream, "Error: Renderer Blocks has to be a Dson::Dict type");
		if (blocks == NULL) return false;
		int blockCutPerCpuThread = CONTEXT blockConfiguration.blockCutPerCpuThread();
		int blockCutPerGpuSM = CONTEXT blockConfiguration.blockCutPerGpuSM();
		bool forceDeviceInstanceUpdate = CONTEXT blockConfiguration.forceDeviceInstanceUpdate();
		if (blocks->contains("cpu_cut")) {
			const Dson::Number *cpuCut = blocks->get("cpu_cut").safeConvert<Dson::Number>(errorStream, "Error: Renderer Blocks CPU_CUT has to be a number");
			if (cpuCut == NULL) return false;
			blockCutPerCpuThread = ((cpuCut->intValue() >= 1) ? cpuCut->intValue() : 1);
		}
		if (blocks->contains("gpu_cut")) {
			const Dson::Number *gpuCut = blocks->get("gpu_cut").safeConvert<Dson::Number>(errorStream, "Error: Renderer Blocks GPU_CUT has to be a number");
			if (gpuCut == NULL) return false;
			blockCutPerGpuSM = ((gpuCut->intValue() >= 1) ? gpuCut->intValue() : 1);
		}
		if (blocks->contains("force_host_block_synchronisation")) {
			const Dson::Bool *forceSynch = blocks->get("force_host_block_synchronisation").safeConvert<Dson::Bool>(errorStream, "Error: Renderer Blocks force_host_block_synchronisation has to be a boolean");
			if (forceSynch == NULL) return false;
			forceDeviceInstanceUpdate = forceSynch->value();
		}
		CONTEXT blockConfiguration = BlockRenderer::BlockConfiguration(blockCutPerCpuThread, blockCutPerGpuSM, forceDeviceInstanceUpdate);
	}

	if (renderer->contains("pixel")) {
		const Dson::Dict *pixel = renderer->get("pixel").safeConvert<Dson::Dict>(errorStream, "Error: Renderer Pixel has to be a Dson::Dict type");
		if (pixel == NULL) return false;
		if (pixel->contains("boxing")) {
			const Dson::String *boxing = pixel->get("boxing").safeConvert<Dson::String>(errorStream, "Error: Renderer Pixel boxing type has to be a string");
			if (boxing == NULL) return false;
			const std::string &boxingText = boxing->value();
			if (boxingText == "height") CONTEXT rendererSettings.boxingMode = DumbRenderer::BOXING_MODE_HEIGHT_BASED;
			else if (boxingText == "width") CONTEXT rendererSettings.boxingMode = DumbRenderer::BOXING_MODE_WIDTH_BASED;
			else if (boxingText == "min") CONTEXT rendererSettings.boxingMode = DumbRenderer::BOXING_MODE_MIN_BASED;
			else if (boxingText == "max") CONTEXT rendererSettings.boxingMode = DumbRenderer::BOXING_MODE_MAX_BASED;
			else {
				if (errorStream != NULL) (*errorStream) << "Error: Renderer Pixel boxing mode can only be one of ['height']/'width'/'min'/'max'" << std::endl;
				return false;
			}
		}
		if (pixel->contains("bounces")) {
			const Dson::Number *bounces = pixel->get("bounces").safeConvert<Dson::Number>(errorStream, "Error: Renderer Pixel bounces has to be a number");
			if (bounces == NULL) return false;
			CONTEXT rendererSettings.maxBounces = bounces->intValue();
		}
		if (pixel->contains("samples_per_pixel")) {
			const Dson::Array *bounces = pixel->get("samples_per_pixel").safeConvert<Dson::Array>(errorStream, "Error: Renderer Pixel samples_per_pixel has to be an array, containing two numbers");
			if (bounces == NULL) return false;
			if (bounces->size() != 2 || bounces->get(0).type() != Dson::Object::DSON_NUMBER || bounces->get(1).type() != Dson::Object::DSON_NUMBER) {
				if (errorStream != NULL) (*errorStream) << "Error: Renderer Pixel samples_per_pixel has to be an array, containing two numbers" << std::endl;
				return false;
			}
			CONTEXT rendererSettings.samplesPerPixelX = ((const Dson::Number*)(&bounces->get(0)))->intValue();
			CONTEXT rendererSettings.samplesPerPixelY = ((const Dson::Number*)(&bounces->get(1)))->intValue();
		}
		if (pixel->contains("pixels_per_gpu_thread")) {
			const Dson::Number *pixelsPerGpuThread = pixel->get("pixels_per_gpu_thread").safeConvert<Dson::Number>(errorStream, "Error: Renderer Pixel pixels_per_gpu_thread has to be a number");
			if (pixelsPerGpuThread == NULL) return false;
			CONTEXT rendererSettings.pixelsPerGPUThread = pixelsPerGpuThread->intValue();
		}
		if (pixel->contains("ignore_backfaces")) {
			const Dson::Bool *ignoreBackfacesObject = pixel->get("ignore_backfaces").safeConvert<Dson::Bool>(errorStream, "Error: Renderer Pixel ignore_backfaces has to have a boolean value");
			if (ignoreBackfacesObject == NULL) return false;
			CONTEXT rendererSettings.ignoreBackfaces = ignoreBackfacesObject->value();
		}
	}

	return true;
}
bool DumbRenderContextData::includeFiles(const Dson::Object &object, std::ostream *errorStream) {
	const Dson::Array *include = object.safeConvert<Dson::Array>(errorStream, "Error: Include should be a list of strings");
	if (include == NULL) return false;
	for (size_t i = 0; i < include->size(); i++) {
		const Dson::String *text = include->get(i).safeConvert<Dson::String>(errorStream, "Error: Can not include non-string objects");
		if (text == NULL) std::cout << std::endl;
		const std::string &fileName = text->value();
		std::string filePath;
		{
			filePath = (CONTEXT sourcePath + fileName);
			std::ifstream stream;
			stream.open(filePath);
			if (stream.fail()) filePath = fileName;
		}
		if (!owner->fromFile(filePath, errorStream)) return false;
	}
	return true;
}


bool DumbRenderContextData::parseMaterial(const Dson::Object &object, std::ostream *errorStream, int *materialId) {
	if (object.type() != Dson::Object::DSON_DICT) {
		if (errorStream != NULL) (*errorStream) << "Error: Material should be contained in Dson::Dict" << std::endl;
		return false;
	}
	const Dson::Dict &dict = (*((Dson::Dict*)(&object)));
	if (!dict.contains("type")) {
		if (errorStream != NULL) (*errorStream) << "Error: Material has to have a type" << std::endl;
		return false;
	}
	else {
		const Dson::Object &typeObject = dict.get("type");
		if (typeObject.type() != Dson::Object::DSON_STRING) {
			if (errorStream != NULL) (*errorStream) << "Error: Material type has to be a string" << std::endl;
			return false;
		}
		const std::string &type = ((Dson::String*)(&typeObject))->value();
		MaterialParserMap::const_iterator it = materialParsers.find(type);
		if (it == materialParsers.end()) {
			if (errorStream != NULL) (*errorStream) << "Error: Unknown material type: \"" << type << "\"" << std::endl;
			return false;
		}
		Material<BakedTriFace> material;
		if (!it->second(material, dict, errorStream, this->owner)) return false;
		if (materialId != NULL) (*materialId) = CONTEXT scene.materials.cpuHandle()->size();
		CONTEXT scene.materials.cpuHandle()->push(material);
	}
	if (dict.contains("name")) {
		const Dson::Object &entry = dict.get("name");
		if (entry.type() == Dson::Object::DSON_STRING)
			CONTEXT materials[((Dson::String*)(&entry))->value()] = (CONTEXT scene.materials.cpuHandle()->size() - 1);
	}
	return true;
}
bool DumbRenderContextData::parseLight(const Dson::Object &object, std::ostream *errorStream) {
	if (object.type() != Dson::Object::DSON_DICT) {
		if (errorStream != NULL) (*errorStream) << "Error: Light should be contained in Dson::Dict" << std::endl;
		return false;
	}
	const Dson::Dict &dict = (*((Dson::Dict*)(&object)));
	if (!dict.contains("type")) {
		if (errorStream != NULL) (*errorStream) << "Error: Light has to have a type" << std::endl;
		return false;
	}
	else {
		const Dson::Object &typeObject = dict.get("type");
		if (typeObject.type() != Dson::Object::DSON_STRING) {
			if (errorStream != NULL) (*errorStream) << "Error: Light type has to be a string" << std::endl;
			return false;
		}
		const std::string &type = ((Dson::String*)(&typeObject))->value();
		LightParserMap::const_iterator it = lightParsers.find(type);
		if (it == lightParsers.end()) {
			if (errorStream != NULL) (*errorStream) << "Error: Unknown light type: \"" << type << "\"" << std::endl;
			return false;
		}
		Light light;
		if (!it->second(light, dict, errorStream, this->owner)) return false;
		CONTEXT scene.lights.cpuHandle()->push(light);
	}
	return true;
}
bool DumbRenderContextData::loadObjFile(const std::string &filename, std::ostream *errorStream) {
	if (CONTEXT objectFiles.find("obj::" + filename) == CONTEXT objectFiles.end()) {
		Stacktor<PolyMesh> meshes;
		Stacktor<String> names;
		std::string objFilePath;
		{
			std::ifstream stream;
			const std::string relativePath = (CONTEXT sourcePath + filename);
			stream.open(relativePath);
			if (!stream.fail()) objFilePath = relativePath;
			else objFilePath = filename;
		}
		if (!MeshReader::readObj(meshes, names, objFilePath)) {
			if (errorStream != NULL) (*errorStream) << ("Error: Could not read file: '" + filename + "' (" + objFilePath + ")") << std::endl;
			return false;
		}
		if (meshes.size() != names.size()) {
			if (errorStream != NULL) (*errorStream) << ("Error: File '" + filename + "' does not have equal amount of names and meshes") << std::endl;
			return false;
		}
		CONTEXT objectFiles["obj::" + filename] = MeshDict();
		MeshDict &polyMeshDict = CONTEXT objectFiles["obj::" + filename];
		for (int i = 0; i < meshes.size(); i++)
			polyMeshDict[names[i] + 0] = meshes[i];
	}
	return true;
}

bool DumbRenderContextData::parseObject(const Dson::Object &object, std::ostream *errorStream) {
	const Dson::Dict *dict = object.safeConvert<Dson::Dict>();
	if (dict != NULL && dict->contains("mesh")) {
		Group::Item item; if (!item.fromDson(&object, errorStream, this)) return false;
		if (item.materialId < 0) {
			if (errorStream != NULL) (*errorStream) << "Error: Object MUST have a material attached to it\n";
			return false;
		}
		scene.geometry.cpuHandle()->push(item.resolve(this));
	}
	else {
		Group::SubGroup group; if (!group.fromDson(&object, errorStream, this)) return false;
		Stacktor<DumbRenderer::SceneType::GeometryUnit> resolved = group.resolve(this);
		for (int i = 0; i < resolved.size(); i++) if (resolved[i].materialId < 0) {
			if (errorStream != NULL) (*errorStream) << "Error: Resolved group must have a material attached to it\n";
			return false;
		}
		scene.geometry.cpuHandle()->push(resolved);
	}
	return true;
}
bool DumbRenderContextData::getMaterialId(const Dson::Object &object, std::ostream *errorStream, int &materialId) {
	if (object.type() == Dson::Object::DSON_STRING) {
		const std::string &name = ((Dson::String*)(&object))->value();
		std::unordered_map<std::string, int>::const_iterator it = materials.find(name);
		if (it == materials.end()) {
			if (errorStream != NULL) (*errorStream) << ("Error: Material not found: \"" + name + "\"") << std::endl;
			return false;
		}
		else materialId = it->second;
	}
	else if (!parseMaterial(object, errorStream, &materialId)) return false;
	return true;
}


namespace {
	class IterationObserver {
	private:
		const FrameBuffer *buffer;
		int lastWidth, lastHeight;
		volatile unsigned int iterationCount, lastIterationCount;
		clock_t startTime, lastTime;
		size_t lastCommentLength;

	public:
		inline IterationObserver(const FrameBuffer *frameBuffer) {
			buffer = frameBuffer;
			buffer->getSize(&lastWidth, &lastHeight);
			iterationCount = lastIterationCount = 0;
			startTime = lastTime = clock();
			lastCommentLength = 0;
		}

		inline void iterationComplete() {
			int width, height; buffer->getSize(&width, &height);
			if ((width != lastWidth) || (height != lastHeight)) {
				lastWidth = width;
				lastHeight = height;
				iterationCount = lastIterationCount = 0;
				std::cout << "\r"; for (size_t i = 0; i < lastCommentLength; i++) std::cout << " ";
			}
			iterationCount++;
			clock_t now = clock();
			clock_t delta = (now - lastTime);
			if (delta >= CLOCKS_PER_SEC) {
				std::cout << "\r"; for (size_t i = 0; i < lastCommentLength; i++) std::cout << " ";
				std::stringstream stream;
				long long seconds = ((long long)(((double)(now - startTime)) / CLOCKS_PER_SEC));
				long long minutes = (seconds / 60); seconds = (seconds - (minutes * 60));
				long long hours = (minutes / 60); minutes = (minutes - (hours * 60));
				stream << std::setprecision(4) << "Iterations:" << iterationCount
					<< " (elapsed: " << hours << ":" << minutes << ":" << seconds
					<< "; Avg Iter/Sec: " << (((double)iterationCount) / ((double)(now - startTime)) * CLOCKS_PER_SEC)
					<< "; Iter/Sec:" << (((double)(iterationCount - lastIterationCount)) / ((double)delta) * CLOCKS_PER_SEC) << ")";
				const std::string &text = stream.str();
				std::cout << "\r" << text;
				lastCommentLength = text.length();
				lastTime = now;
				lastIterationCount = iterationCount;
			}
		}

		inline static void iterationCompleteCallback(void *observer) {
			((IterationObserver*)observer)->iterationComplete();
		}
	};
}



void DumbRenderContext::runWindowRender() {
	CONTEXT scene.geometry.cpuHandle()->build();
	CONTEXT scene.geometry.makeDirty();

	std::cout << "_____________________________________________________________" << std::endl;
	std::cout << "RENDERING:" << std::endl;
	std::cout << "    ________________________________________" << std::endl;
	std::cout << "    CPU threads: " << CONTEXT threadConfiguration.numHostThreads() << std::endl;
	if (CONTEXT threadConfiguration.numDevices() > 0) {
		std::cout << "    GPU threads: ";
		if (CONTEXT threadConfiguration.numDevices() == 1) std::cout << CONTEXT threadConfiguration.numDeviceThreads(0) << " [" << Device::getDeviceName(0) << "]" << std::endl;
		else {
			std::cout << std::endl;
			for (int i = 0; i < CONTEXT threadConfiguration.numDevices(); i++)
				std::cout << "        GPU " << i << ": " << CONTEXT threadConfiguration.numDeviceThreads(i) << " [" << Device::getDeviceName(i) << "]" << std::endl;
		}
	}
	std::cout << "    Block cut per CPU thread: " << CONTEXT blockConfiguration.blockCutPerCpuThread() << std::endl;
	std::cout << "    Block cut per GPU SM:     " << CONTEXT blockConfiguration.blockCutPerGpuSM() << std::endl;
	std::cout << "    ________________________________________" << std::endl;
	std::cout << "    Geometry:    " << CONTEXT scene.geometry.cpuHandle()->getData().size() << " tris" << std::endl;
	std::cout << "    Node count:  " << CONTEXT scene.geometry.cpuHandle()->getNodeCount() << std::endl;
	std::cout << "    Materials:   " << CONTEXT scene.materials.cpuHandle()->size() << std::endl;
	std::cout << "    Lights:      " << CONTEXT scene.lights.cpuHandle()->size() << std::endl;
	std::cout << "    Bounces:     " << CONTEXT rendererSettings.maxBounces << std::endl;
	std::cout << "_____________________________________________________________" << std::endl;

	FrameBufferManager frameBuffer;
	frameBuffer.cpuHandle()->use<BlockBuffer>();

	DumbRenderer renderer(CONTEXT threadConfiguration, CONTEXT blockConfiguration,
		&frameBuffer, &CONTEXT scene, &CONTEXT camera,
		CONTEXT rendererSettings.boxingMode, CONTEXT rendererSettings.maxBounces,
		CONTEXT rendererSettings.samplesPerPixelX, CONTEXT rendererSettings.samplesPerPixelY,
		CONTEXT rendererSettings.pixelsPerGPUThread, CONTEXT rendererSettings.ignoreBackfaces);

	int renderingDevice = 0;
	for (int i = 0; i < CONTEXT threadConfiguration.numDevices(); i++)
		if (CONTEXT threadConfiguration.numDeviceThreads(i) > 0) {
			renderingDevice = i;
			break;
		}
	BufferedWindow bufferedWindow(renderer.automaticallySynchesHostBlocks() ? 0 : BufferedWindow::SYNCH_FRAME_BUFFER_FROM_DEVICE, NULL, L"Render Viewport", NULL, renderingDevice);

	BufferedRenderProcess process;
	process.setBuffer(&frameBuffer);
	process.setInfinateTargetIterations();
	process.setTargetResolutionToWindowSize();
	process.setTargetDisplayWindow(&bufferedWindow);
	process.setRenderer(&renderer);

	IterationObserver observer(frameBuffer.cpuHandle());
	process.setIterationCompletionCallback(IterationObserver::iterationCompleteCallback, &observer);

	process.start();
#ifndef _WIN32
	int sleeps = 0;
#endif	
	while (!bufferedWindow.windowClosed()) {
		std::this_thread::sleep_for(std::chrono::milliseconds(32));
#ifndef _WIN32
		sleeps++; if (sleeps % 512 == 0) Images::saveBufferPNG(*frameBuffer.cpuHandle(), "test_image.png");
#endif
	}
	process.end();
	{
		std::cout << std::endl << "Enter a name ending with '.png' to save the image: ";
		std::string line; std::getline(std::cin, line);
		size_t leadInWhiteSpaces; for (leadInWhiteSpaces = 0; leadInWhiteSpaces < line.length(); leadInWhiteSpaces++) 
			if (!iswspace(line[leadInWhiteSpaces])) break;
		size_t leadOutWhiteSpaces; for (leadOutWhiteSpaces = 0; leadOutWhiteSpaces < line.length(); leadOutWhiteSpaces++) 
			if (!iswspace(line[line.length() - leadOutWhiteSpaces - 1])) break;
		if (leadInWhiteSpaces < line.length()) {
			std::string filename = line.substr(leadInWhiteSpaces, line.length() - leadInWhiteSpaces - leadOutWhiteSpaces);
			if (filename.length() >= 4) if (filename.substr(filename.length() - 4, 4) == ".png") {
				std::cout << "Saving...." << std::endl;
				if (Images::saveBufferPNG(*frameBuffer.cpuHandle(), filename) == Images::IMAGES_NO_ERROR)
					std::cout << ("Image saved at: '" + filename + "'...") << std::endl;
				else std::cout << ("Failed to save image at: '" + filename + "'...") << std::endl;
			}
		}
	}
}


void DumbRenderContext::test() {
	std::cout << "Enter scene file name: ";
	std::string filename;
	std::getline(std::cin, filename);
	testFile(filename);
}
void DumbRenderContext::testFile(const std::string &filename) {
	DumbRenderContext context;
	if (!context.fromFile(filename, &std::cout)) {
		std::string line;
		std::getline(std::cin, line);
	}
	context.runWindowRender();
}




namespace {
	struct RenderInstanceData {
		FrameBufferManager frameBuffer;
		DumbRenderer renderer;
		BufferedRenderProcess process;
		BufferedWindow bufferedWindow;
		Window *window;

		RenderInstanceData(Window *wnd) : bufferedWindow(0, wnd) {
			window = wnd;
		}

		std::mutex lock;
		std::set<std::pair<DumbRenderContext::RenderInstance::Callback, void *> > onIterationComplete;
	};

#define DATA_PTR ((RenderInstanceData*)data)
#define DATA DATA_PTR->
#define CTX ((DumbRenderContextData*)ctx->data)->
}


DumbRenderContext::RenderInstance::RenderInstance(DumbRenderContext *context, Window *window) {
	ctx = context;
	data = new RenderInstanceData(window);
	reset();
}
DumbRenderContext::RenderInstance::~RenderInstance() {
	DATA process.lockSettings();
	DATA renderer.interruptRender();
	DATA process.unlockSettings();
	stop();
	RenderInstanceData* instanceData = DATA_PTR;
	if (instanceData != NULL) {
		delete instanceData;
		data = NULL;
	}
}

void DumbRenderContext::RenderInstance::interruptRender() { 
	DATA process.lockSettings();
	DATA renderer.interruptRender();
	DATA process.unlockSettings();
}
void DumbRenderContext::RenderInstance::uninterruptRender() { 
	DATA process.lockSettings();
	DATA renderer.uninterruptRender();
	DATA process.unlockSettings();
}
bool DumbRenderContext::RenderInstance::renderInterrupted()const { return DATA renderer.renderInterrupted(); }

void DumbRenderContext::RenderInstance::reset() {
	initBuffer();
	initRenderer();
	initWindow();
	initRenderContext();
}

void DumbRenderContext::RenderInstance::start() {
	DATA process.start();
}
void DumbRenderContext::RenderInstance::stop() {
	DATA process.end();
}

void DumbRenderContext::RenderInstance::setResolution(int width, int height) {
	DATA process.setTargetResolution(width, height, true);
}
void DumbRenderContext::RenderInstance::getResolution(int &width, int &height) {
	DATA frameBuffer.cpuHandle()->getSize(&width, &height);
}
void DumbRenderContext::RenderInstance::getPixelColor(int x, int y, float &r, float &g, float &b, float &a)const {
	Color color = DATA frameBuffer.cpuHandle()->getColor(x, y);
	r = color.r; g = color.g; b = color.b; a = color.a;
}

void DumbRenderContext::RenderInstance::onIterationComplete(Callback callback, void *aux) {
	DATA lock.lock();
	DATA onIterationComplete.emplace(std::pair<DumbRenderContext::RenderInstance::Callback, void *>(callback, aux));
	DATA lock.unlock();
}

int DumbRenderContext::RenderInstance::iteration()const {
	return DATA renderer.iteration();
}
double DumbRenderContext::RenderInstance::renderTime()const {
	return (((double)DATA process.renderTime()) / CLOCKS_PER_SEC);
}

int DumbRenderContext::RenderInstance::cpuThreads()const { return DATA renderer.threadConfiguration().numHostThreads(); }
void DumbRenderContext::RenderInstance::setCpuThreads(int count) { DATA renderer.threadConfiguration().configureCPU(count); }

int DumbRenderContext::RenderInstance::gpuCount()const { return DATA renderer.threadConfiguration().numDevices(); }
bool DumbRenderContext::RenderInstance::gpuOn(int index)const { return (DATA renderer.threadConfiguration().numDeviceThreads(index) > 0); }
void DumbRenderContext::RenderInstance::setGpu(int index, bool on) { DATA renderer.threadConfiguration().configureGPU(index, on ? CTX threadConfiguration.numDeviceThreads(index) : 0); }





void DumbRenderContext::RenderInstance::initBuffer() {
	DATA frameBuffer.~FrameBufferManager();
	new (&DATA frameBuffer) FrameBufferManager();
	DATA frameBuffer.cpuHandle()->use<BlockBuffer>();
}
void DumbRenderContext::RenderInstance::initRenderer() {
	DATA renderer.~DumbRenderer();
	Renderer::ThreadConfiguration threadConfiguration = CTX threadConfiguration;
	int threadsOnCpu = threadConfiguration.numHostThreads();
	threadConfiguration.configureCPU(Renderer::ThreadConfiguration::ALL);
	new (&DATA renderer) DumbRenderer(threadConfiguration, CTX blockConfiguration,
		&DATA frameBuffer, &CTX scene, &CTX camera,
		CTX rendererSettings.boxingMode, CTX rendererSettings.maxBounces,
		CTX rendererSettings.samplesPerPixelX, CTX rendererSettings.samplesPerPixelY,
		CTX rendererSettings.pixelsPerGPUThread, CTX rendererSettings.ignoreBackfaces);
	DATA renderer.threadConfiguration().configureCPU(threadsOnCpu);
}
void DumbRenderContext::RenderInstance::initWindow() {
	DATA bufferedWindow.~BufferedWindow();
	int renderingDevice = 0;
	for (int i = 0; i < CTX threadConfiguration.numDevices(); i++)
		if (CTX threadConfiguration.numDeviceThreads(i) > 0) {
			renderingDevice = i;
			break;
		}
	new (&DATA bufferedWindow) BufferedWindow(
		DATA renderer.automaticallySynchesHostBlocks() ? 0 : BufferedWindow::SYNCH_FRAME_BUFFER_FROM_DEVICE, 
		DATA window, L"Render Viewport", NULL, renderingDevice);
}
void DumbRenderContext::RenderInstance::initRenderContext() {
	DATA process.lockSettings();
	DATA process.setBuffer(&DATA frameBuffer, false);
	DATA process.setInfinateTargetIterations(false);
	DATA process.setTargetResolutionToWindowSize(false);
	DATA process.setRenderer(&DATA renderer, false);
	DATA process.setTargetDisplayWindow(&DATA bufferedWindow, false);
	DATA process.setIterationCompletionCallback(iterationComplete, this, false);
	DATA process.unlockSettings();
}


void DumbRenderContext::RenderInstance::iterationComplete() {
	for (std::set<std::pair<Callback, void *> >::const_iterator it = DATA onIterationComplete.begin(); it != DATA onIterationComplete.end(); it++)
		it->first(it->second);
}
void DumbRenderContext::RenderInstance::iterationComplete(void *reference) {
	((RenderInstance*)reference)->iterationComplete();
}
