#include "hip/hip_runtime.h"
#include "Renderer.test.cuh"
#include "Renderer.cuh"
#include "../../../Namespaces/Tests/Tests.h"
#include <time.h>



namespace RendererTest {
	namespace {
		class DummyRenderer : public Renderer {
		public:
			DummyRenderer(const ThreadConfiguration &configuration, bool verbose, int counter);
			virtual ~DummyRenderer();

		protected:
			virtual bool setupSharedData(const Info &info, void *& sharedData);
			virtual bool setupData(const Info &info, void *& data);
			virtual bool prepareIteration();
			virtual void iterateCPU(const Info &info);
			virtual void iterateGPU(const Info &info);
			virtual bool completeIteration();
			virtual bool clearData(const Info &info, void *& data);
			virtual bool clearSharedData(const Info &info, void *& sharedData);

		private:
			std::mutex printLock;
			bool dumpCalls;
			int cnt;

		public:
			int *total[128];
		};
		DummyRenderer::DummyRenderer(const ThreadConfiguration &configuration, bool verbose, int counter) : Renderer(configuration){
			dumpCalls = verbose;
			cnt = counter;
			for (int i = 0; i < 128; i++) total[i] = NULL;
		}
		DummyRenderer::~DummyRenderer() { 
			killRenderThreads(); 
		}
		bool DummyRenderer::setupSharedData(const Info &info, void *& sharedData) {
			if (dumpCalls) {
				printLock.lock();
				std::cout << "setupSharedData - ";
				if (info.isGPU()) std::cout << "GPU " << info.device << " ";
				else std::cout << "CPU ";
				std::cout << info.deviceThreadId << std::endl;
				printLock.unlock();
			}
			sharedData = (void*)(&info.device);
			return true; 
		}
		bool DummyRenderer::setupData(const Info &info, void *& data) {
			if (*(((int*)info.sharedData)) != info.device) std::cout << "ERROR: sharedData VALUE INCORRECT" << std::endl;
			data = (void*)(&info.deviceThreadId);
			if (dumpCalls) {
				printLock.lock(); 
				std::cout << "setupData - ";
				if (info.isGPU()) std::cout << "GPU " << info.device << " ";
				else std::cout << "CPU ";
				std::cout << info.deviceThreadId << std::endl;
				printLock.unlock();
			}
			total[info.globalThreadId] = new int;
			return true; 
		}
		bool DummyRenderer::prepareIteration() {
			if (dumpCalls) {
				printLock.lock();
				std::cout << "prepareIteration..." << std::endl;
				printLock.unlock();
			}
			return true;
		}
		void DummyRenderer::iterateCPU(const Info &info) {
			if (*(((int*)info.sharedData)) != info.device) std::cout << "ERROR: sharedData VALUE INCORRECT" << std::endl;
			if (*(((int*)info.data)) != info.deviceThreadId) std::cout << "ERROR: data VALUE INCORRECT" << std::endl;
			if (dumpCalls) {
				printLock.lock();
				std::cout << "iterateCPU - ";
				if (info.isGPU()) std::cout << "GPU " << info.device << " ";
				else std::cout << "CPU ";
				std::cout << info.deviceThreadId << std::endl;
				printLock.unlock();
			}
			int &v = (*total[info.globalThreadId]);
			for (int i = 0; i < cnt; i++) v++;
		}
		void DummyRenderer::iterateGPU(const Info &info) {
			if (*(((int*)info.sharedData)) != info.device) std::cout << "ERROR: sharedData VALUE INCORRECT" << std::endl;
			if (*(((int*)info.data)) != info.deviceThreadId) std::cout << "ERROR: data VALUE INCORRECT" << std::endl;
			if (dumpCalls) {
				printLock.lock();
				std::cout << "iterateGPU - ";
				if (info.isGPU()) std::cout << "GPU " << info.device << " ";
				else std::cout << "CPU ";
				std::cout << info.deviceThreadId << std::endl;
				printLock.unlock();
			}
			int &v = (*total[info.globalThreadId]);
			for (int i = 0; i < cnt; i++) v++;
		}
		bool DummyRenderer::completeIteration() {
			if (dumpCalls) {
				printLock.lock();
				std::cout << "completeIteration..." << std::endl;
				printLock.unlock();
			}
			return true;
		}
		bool DummyRenderer::clearData(const Info &info, void *& data) {
			if (*(((int*)info.sharedData)) != info.device) std::cout << "ERROR: sharedData VALUE INCORRECT" << std::endl;
			data = NULL;
			if (dumpCalls) {
				printLock.lock();
				std::cout << "clearData - ";
				if (info.isGPU()) std::cout << "GPU " << info.device << " ";
				else std::cout << "CPU ";
				std::cout << info.deviceThreadId << std::endl;
				printLock.unlock();
			}
			delete total[info.globalThreadId];
			return true; 
		}
		bool DummyRenderer::clearSharedData(const Info &info, void *& sharedData) {
			if (*(((int*)info.sharedData)) != info.device) std::cout << "ERROR: sharedData VALUE INCORRECT" << std::endl;
			sharedData = NULL;
			if (info.data != NULL) std::cout << "ERROR: data NOT NULL";
			if (dumpCalls) {
				printLock.lock();
				std::cout << "clearSharedData - ";
				if (info.isGPU()) std::cout << "GPU " << info.device << " ";
				else std::cout << "CPU ";
				std::cout << info.deviceThreadId << std::endl;
				printLock.unlock();
			}
			return true; 
		}

		void makeAndDestroy() {
			std::cout << std::endl << "--> FAST CREATE & DESTROY TEST: " << std::endl;
			Renderer::ThreadConfiguration configuration;
			DummyRenderer renderer(configuration, true, 0);
			renderer.iterate();
		}

		void testIterationsSpeed() {
			const int n = 8192;
			const int weight = 262144;
			Renderer::ThreadConfiguration configuration;
			DummyRenderer renderer(configuration, false, weight);
			std::cout << std::endl << "--> RUNNING " << n << " ITERATIONS (weight "<< weight << ")...." << std::endl;
			long t = clock();
			for (int i = 0; i < n; i++) renderer.iterate();
			long deltaTime = (clock() - t);
			float secs = (((float)deltaTime) / CLOCKS_PER_SEC);
			std::cout << "TIME: " << deltaTime << " CLOCK TICKS (" << secs << " sec)" << std::endl;
			std::cout << "SPEED: " << (((float)n) / secs) << " ITERATIONS PER SECOND" << std::endl << std::endl;
			int total = 0;
			for (int i = 0; i < 128; i++) 
				if(renderer.total[i] != NULL) total += (*renderer.total[i]);
			std::cout << "TOTAL: " << total << std::endl;
		}

		void runTest() {
			makeAndDestroy();
			testIterationsSpeed();
		}

		void testRenderer(){
			while (true) {
				std::cout << "Enter anthing to run basic Renderer template test: ";
				std::string s;
				std::getline(std::cin, s);
				if (s.length() <= 0) break;
				runTest();
			}
		}
	}

	void test() {
		Tests::runTest(testRenderer, "Testing basic Renderer template");
		hipSetDevice(0);
	}
}

